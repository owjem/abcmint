#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdint.h>
#include <stdio.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>
#include <inttypes.h>

#define K PARK

#define NUM_THREADS (1 << (N-K))
#define BLOCK_DIM (NUM_THREADS > 128 ? 128 : NUM_THREADS)
#define GRID_DIM (NUM_THREADS/BLOCK_DIM)

//#define PRINT_SOL(X) printf("%lX\n", X)
#define PRINT_SOL(X)

//#define LOG(level, f_, ...) fprintf(stderr, (f_), ##__VA_ARGS__)
#define LOG(level, f_, ...)


#define COEF(I,J) ((((J)*((J)-1))>>1) + (I))
#if 0
/* for parsing challenge file */
const char* CHA_GF_LINE = "Galois Field";
const char* CHA_VAR_LINE = "Number of variables";
const char* CHA_EQ_LINE = "Number of polynomials";
const char* CHA_SEED_LINE = "Seed";
//const char* CHA_ORD_LINE = "Order";
const char* CHA_EQ_START = "*********";
const size_t MAX_PRE_LEN = 128;
#endif

template <class Type>
class cudaData
{
public:
	cudaData(size_t len_host, size_t len_dev=0);
	~cudaData();

	size_t size_host();
	size_t size_dev();
	void clear();
	void write(size_t off_src=0, size_t size=0, size_t off_des=0);		
	void read(size_t off_src=0, size_t size=0, size_t off_des=0);

	Type *host;
	Type *dev;

private:
	size_t sz_host;
	size_t sz_dev;
};


template <class Type>
cudaData<Type>::cudaData(size_t len_host, size_t len_dev)
{
	if(len_dev == 0) len_dev = len_host;

	sz_host = len_host * sizeof(Type);
	sz_dev  = len_dev * sizeof(Type);

	host = (Type*) malloc(sz_host);	

	hipMalloc((void**) &dev, sz_dev);
}

template <class Type>
cudaData<Type>::~cudaData()
{       if(host)
	free(host);
        if(dev)
	hipFree(dev);
}

template <class Type>
size_t cudaData<Type>::size_host()
{
	return sz_host;
}

template <class Type>
size_t cudaData<Type>::size_dev()
{
	return sz_dev;
}

template <class Type>
void cudaData<Type>::clear()
{
	memset(host, 0, sz_host);
}

template <class Type>
void cudaData<Type>::write(size_t off_src, size_t size, size_t off_des)
{
	if(size == 0) size = (sz_host <= sz_dev) ? sz_host : sz_dev;

	hipMemcpy(&dev[off_src], &host[off_des], size, hipMemcpyHostToDevice);
}

template <class Type>
void cudaData<Type>::read(size_t off_src, size_t size, size_t off_des)
{
	if(size == 0) size = (sz_host <= sz_dev) ? sz_host : sz_dev;

	hipMemcpy(&host[off_src], &dev[off_des], size, hipMemcpyDeviceToHost);
}

#if 0
/* testing if pre is a prefix of the string */
static inline bool
check_prefix(const char* pre, const char* str) {
      return !strncmp(pre, str, strnlen(pre, MAX_PRE_LEN));
}

/* parse the header of challenge file, return true is still in header.
    * return false otherwise.
     */
static bool
parse_cha_header(const char* str, uint32_t &N, uint32_t &M) {
  bool verbose = false;
  if(check_prefix(CHA_EQ_START, str)) {
    if(verbose) {
      printf("\t\treading equations...\n");
    }
    return false;
  }

  uint64_t var_num, eq_num, seed;

  if(check_prefix(CHA_VAR_LINE, str)) {
    if(1 != sscanf(str, "%*s %*s %*s %*s : %" PRIu64, &var_num)) {
      fprintf(stderr, "[!] cannot parse number of unknowns: %s\n", str);
      exit(-1);
    }

    N = var_num;

//    if (var_num != N)
//    {
//      fprintf(stderr, "Number of variables in input file does not fit compile options!\n");
//      fprintf(stderr, "%" PRIu64 " != %i\n", var_num, N);
//      exit(-1);
//    }

    if(verbose) {
      printf("\t\tnumber of variables: %" PRIu64 "\n", var_num);
    }

  } else if(check_prefix(CHA_EQ_LINE, str)) {
    if(1 != sscanf(str, "%*s %*s %*s %*s : %" PRIu64, &eq_num)) {
      fprintf(stderr, "[!] cannot parse number of equations: %s\n", str);
      exit(-1);
    }

    M = eq_num;

//    if (eq_num != M)
//    {
//      fprintf(stderr, "Number of equations in input file does not fit compile options!\n");
//      fprintf(stderr, "%" PRIu64 " != %i\n", eq_num, M);
//      exit(-1);
//    }

    if(verbose) {
      printf("\t\tnumber of equations: %" PRIu64 "\n", eq_num);
    }

  } else if(check_prefix(CHA_SEED_LINE, str)) {
    if(1 != sscanf(str, "%*s : %" PRIu64, &seed)) {
      fprintf(stderr, "[!] unable to seed: %s\n", str);
      exit(-1);
    }

    if(verbose) {
      printf("\t\tseed: %" PRIu64 "\n", seed);
    }

  } else if(check_prefix(CHA_GF_LINE, str)) {
    int prime = 0;
    if( (1 != sscanf(str, "%*s %*s : GF(%d)", &prime)) || prime != 2) {
      fprintf(stderr, "[!] unable to process GF(%d)\n", prime);
      exit(-1);
    }

    if(verbose) {
      printf("\t\tfield: GF(%d)\n", prime);
    }
  }

  return true;
}

/* parse the system of challenge file. Note this will destroy the string */
static void
parse_cha_eqs(char* str, const uint64_t eq_idx, uint32_t *orig_sys, uint32_t N) {
  char* ptr = NULL;

  uint64_t i = 0;
  ptr = strtok(str, " ;");
  while(NULL != ptr) {
    orig_sys[(N*(N-1)/2 + N + N + 1)*eq_idx +i] = atoi(ptr);
    i += 1;
    ptr = strtok(NULL, " ;\n");
  }
}

uint32_t *read_sys(uint32_t &N, uint32_t &M)
{
  FILE* fp = stdin;
  //FILE* fp = fopen( "data.in" , "r");

  // NOTE: expand the buffer if needed
  const size_t buf_size = 0x1 << 20; // 1MB per line
  char* buf = (char*)malloc(buf_size);
  uint64_t eq_idx = 0;

  while (NULL != fgets(buf, buf_size, fp)) {
    if (!parse_cha_header(buf, N, M))
      break;
  }

  if (feof(fp))
    return NULL;

  uint32_t* data = (uint32_t*)malloc((N*(N-1)/2 + 2*N + 1)*M*sizeof(uint32_t));

  for (int i = 0; i < M; i++) {
    if (NULL != fgets(buf, buf_size, fp)) {
      parse_cha_eqs(buf, eq_idx++, data, N);
    }
    else
    {
      free(buf);
      free(data);

      fprintf(stderr, "Error while reading input data!\n");
      exit(-1);
    }
  }

  if (feof(fp))
    fprintf(stderr, "end of file\n");

  //fclose(fp);
  free(buf);

  return data;
}

#endif

#ifndef HAVE_CNT
static const int MultiplyDeBruijnBitPosition[32] = 
{
    0, 1, 28, 2, 29, 14, 24, 3, 30, 22, 20, 15, 25, 17, 4, 8, 
      31, 27, 13, 23, 21, 19, 16, 7, 26, 12, 18, 6, 11, 5, 10, 9
};

static uint32_t cnt0(uint32_t v)
{
  return MultiplyDeBruijnBitPosition[((uint32_t)((v & -v) * 0x077CB531U)) >> 27];
}
#define HAVE_CNT
#endif


uint32_t* check_thread(uint32_t *deg2, uint32_t *deg1, uint32_t thread, uint32_t N)
{
  uint32_t rounds;
  uint32_t x = 0;
  uint32_t y = 0;
  uint32_t z = 0;
  uint32_t tmp = 0;
  uint32_t count = 0;

  uint32_t diff[K];

  uint32_t* result = (uint32_t*) malloc( (1 << K) * sizeof(uint32_t) );

  diff[0] = deg1[0 * NUM_THREADS + thread];

  for (int i = 1; i < K; i++)
  {
    diff[i] = deg1[i * NUM_THREADS + thread] ^ deg2[COEF(i-1, i)];
  }

  uint32_t res = deg1[K * NUM_THREADS + thread];

  for( rounds = 1; rounds < (1 << K); rounds += 1)
  {
    tmp = (rounds & (rounds-1));
    y = rounds ^ tmp;
    x ^= y;
    z = tmp ^ (tmp & (tmp-1));

    uint32_t y_pos = cnt0(y);
    uint32_t z_pos = cnt0(z);

    if (z_pos > y_pos)
      diff[y_pos] ^= deg2[COEF(y_pos, z_pos)];

    res ^= diff[y_pos];
    if( res == 0 ) result[ count++ ] = x;
  }

  result[ count ] = 0;

  return result;
}


void deg0_coefs(uint32_t *deg2, uint32_t *deg1, uint32_t *result, uint32_t N)
{
	uint32_t rounds;
	uint32_t x = 0;
	uint32_t y = 0;
	uint32_t z = 0;
	uint32_t tmp = 0;

  uint32_t diff[N-K];

	diff[0] = deg1[0];

  for (int i = 1; i < (N-K); i++)
  {
    diff[i] = deg1[i] ^ deg2[COEF(i-1, i)];
  }

	uint32_t res = deg1[N-K];

	result[0] = res;

	for (rounds = 1; rounds < (1 << (N-K)); rounds += 1)
	{
		tmp = (rounds & (rounds-1));
		y = rounds ^ tmp;
		x ^= y;
		z = tmp ^ (tmp & (tmp-1));

    uint32_t y_pos = cnt0(y);
    uint32_t z_pos = cnt0(z);

    if (z_pos > y_pos)
      diff[y_pos] ^= deg2[COEF(y_pos, z_pos)];

    res ^= diff[y_pos];
    result[ x ] = res;
    tmp = (y_pos * (y_pos-1)) >> 1;
	}
}

void deg1_coefs(uint32_t *deg1, uint32_t *result, uint32_t N)
{
	uint32_t x = 0;
	uint32_t y = 0;
	uint32_t res = deg1[N-K];

	result[0] = res;

	for(uint32_t rounds = 1; rounds < (1 << (N-K)); rounds += 1)
	{
		y = rounds ^ (rounds & (rounds-1));
		x ^= y;

    res ^= deg1[cnt0(y)];
    result[ x ] = res;
	}
}

void partial_eval(uint32_t *sys, uint32_t *deg1, uint32_t N)
{
	uint32_t deg1_sys[(N-K)+1];
	uint32_t deg2_sys[COEF(N-K, N-K)+1];
	uint64_t pos = 0;

	// deg2 part
	for(uint32_t i = 0; i < K; i++)
	{
    for(uint32_t j = 0; j <= (N-K); j++)
    {
      deg1_sys[j] = sys[COEF(0, j + K) + i];
    }

		deg1_coefs(deg1_sys, &deg1[pos], N);
		pos += (1 << (N-K));
	}

	// deg1 part
  for(uint32_t j = 1; j <= (N-K); j++)
  {
    for(uint32_t i = 0; i <= j; i++)
    {
      deg2_sys[COEF(i, j)] = sys[COEF(i + K, j + K)];
    }
  }

  deg0_coefs(deg2_sys, deg2_sys + COEF(0, N-K), &deg1[pos], N);
}



uint32_t check_sol(uint32_t *sys, uint64_t sol, uint32_t N, uint32_t M)
{
	uint32_t i, j, pos = 0;
	uint32_t x[N], check = 0;
   
  for (uint32_t b = 0; b < M; b +=32)
  {
    uint32_t mask = (M-b) >= 32 ? 0xffffffff : ((1 << (M-b))-1);

    for (i = 0; i < N; i++)
      x[i] = ((sol >> i) & 1) ? mask : 0;

    // computing quadratic part
    for(j = 1; j < N; j++)
      for(i = 0; i < j; i++) 
        check ^= sys[pos++] & x[i] & x[j];

    // computing linear part
    for(i = 0; i < N; i++) 
      check ^= sys[pos++] & x[i];

    // constant part
    check ^= sys[pos++];
  }

	return check;
}



double get_ms_time(void) {
	struct timeval timev;

	gettimeofday(&timev, NULL);
	return (double) timev.tv_sec * 1000 + (double) timev.tv_usec / 1000;
}

extern uint32_t check_sol(uint32_t *sys, uint64_t sol, uint32_t N, uint32_t M);
extern void partial_eval(uint32_t *sys, uint32_t *deg1, uint32_t N);
extern uint32_t* check_thread(uint32_t *deg2, uint32_t *deg1, uint32_t thread, uint32_t N);

__device__ __constant__ uint32_t deg2_block[ K*(K-1)/2 ];

#include "kernel.inc"

uint32_t *pack_sys_data(uint32_t *data, uint32_t N, uint32_t M)
{
//  reduce input system - remove squares

  uint32_t num_blocks = ((M >> 5) + ((M & 31) == 0 ? 0 : 1));

  uint32_t *sys = (uint32_t*)malloc(sizeof(uint32_t)*(N*(N-1)/2 + N + 1)*num_blocks);
  uint32_t *sq0 = (uint32_t*)malloc(sizeof(uint32_t)*N*num_blocks);


  int sq_id = 0;

  int is = 0;
  int id = 0;
  
  for (int v0 = 0; v0 < N; v0++)
  {
    for (int v1 = 0; v1 <= v0; v1++)
    {
      for (uint32_t b = 0; b < M; b +=32)
      {
        uint32_t val = 0;

        for (int j = (((M - b) >= 32) ? b + 31 : (M-1)); j >= (int)b; j--)
          val = (val << 1) | data[(N*(N-1)/2 + N + N + 1) * j + is];

        if (v0 == v1)
          sq0[sq_id + N*(b >> 5)] = val;
        else
          sys[(N*(N-1)/2 + N + 1) * (b >> 5) + id] = val;
      }

      is += 1;

      if (v0 == v1)
        sq_id += 1;
      else
        id += 1;
    }
  }

  for (int v0 = 0; v0 < N; v0++)
  {
    for (uint32_t b = 0; b < M; b +=32)
    {
      uint32_t val = 0;

      for (int j = (((M - b) >= 32) ? b + 31 : (M-1)); j >= (int)b; j--)
        val = (val << 1) | data[(N*(N-1)/2 + N + N + 1) * j + is];

      sys[(N*(N-1)/2 + N + 1) * (b >> 5) + id] = val ^ sq0[v0 + N*(b >> 5)];
    }

    is += 1;
    id += 1;
  }

  {
    for (uint32_t b = 0; b < M; b +=32)
    {
      uint32_t val = 0;

      for (int j = (((M - b) >= 32) ? b + 31 : (M-1)); j >= (int)b; j--)
        val = (val << 1) | data[(N*(N-1)/2 + N + N + 1) * j + is];

      sys[(N*(N-1)/2 + N + 1) * (b >> 5) + id] = val;
    }
  }
//  if(data)
//  free(data);
  if(sq0)
  free(sq0);
  return sys;
}

int cuda_device = 0;

void setDevice(int device)
{
  cuda_device = device;
}

uint64_t GPUSearchSolution(uint32_t* coefficients, unsigned int number_of_variables, 
                           unsigned int number_of_equations)
{
  static bool init = false;

  if (!init)
  {
  //  double initTime = 0;
  //  initTime -= get_ms_time();

    // set to designated device
    //int test;
    hipSetDevice(cuda_device);
    //hipGetDevice(&test);
    //assert(atoi(argv[1]) == test);

  //  initTime += get_ms_time();
  //  LOG(INFO, "init time = %f\n", initTime);

    init = true;
  }


//  double preTime = 0, memTime = 0, recvTime = 0, checkTime = 0, ctTime = 0;
//  float kernelTime = 0;
  uint32_t solCount = 0, ctCount = 0;

  uint64_t res = 0;

  // create events here
  hipEvent_t start, stop;
  hipEventCreate (&start);
  hipEventCreate (&stop);


  uint32_t N = number_of_variables;
  uint32_t M = number_of_equations;


  if (N <= K)
  {
    fprintf(stderr, "N must be larger than K!\n");
    exit(-1);
  }

  uint32_t *sys = pack_sys_data(coefficients, N, M);

//  preTime -= get_ms_time(); // partial evaluation

  cudaData<uint32_t> deg1((K + 1) * NUM_THREADS);

  partial_eval(sys, deg1.host, N);

//  preTime += get_ms_time();

//  memTime -= get_ms_time(); // initializing GPU memory space

  // initialize constant memory space for the quadratic part
  hipMemcpyToSymbol(HIP_SYMBOL(deg2_block), sys, sizeof(uint32_t) * K*(K-1)/2);

  // initialize global memory space for the linear parts
  deg1.write();

  // initialize global memory space for the results of each threads
  cudaData<uint32_t> result(NUM_THREADS);

 // memTime += get_ms_time(); 

  // launch kernel function and measure the elapsed time
//  hipEventRecord(start, 0);

  guess<<<GRID_DIM, BLOCK_DIM>>>(deg1.dev, result.dev, NUM_THREADS);

//  hipEventRecord(stop, 0);
//  hipEventSynchronize(stop);

//  hipEventElapsedTime(&kernelTime, start, stop);

 // recvTime -= get_ms_time(); // copy the results of each thread to host

  result.read();

//  recvTime += get_ms_time();

//  checkTime -= get_ms_time(); // check if the results are available

  int32_t ans; 

  for(uint64_t i = 0; i < NUM_THREADS; i++)
  {
    ans = result.host[i];

    if(ans)
    {
      solCount++;

      if (ans & 0x80000000) // more than one solution 
      {
        uint32_t *sols;

        ctCount++;
     //   ctTime -= get_ms_time();
        sols = check_thread(sys, deg1.host, i, N);
     //   ctTime += get_ms_time();

        uint32_t j;

        for(j = 0; sols[j]; j++)
        {
          if (check_sol(sys, (i << K) | sols[j], N, M) == 1)
          {
         //   LOG(INFO, "thread %lX ---------> solution %X\n", i, sols[j]);
            PRINT_SOL((i << K) | sols[j]);

            res = (i << K) | sols[j];
            free(sols);
            goto end;
           
          }
        }

       // LOG(INFO, "thread %lX ---------> several solutions: %u\n", i, j);

        free(sols);
      }
      else // only one solution
      {
        if (check_sol(sys, (i << K) | ans, N, M) == 0)
        {
     //     LOG(INFO, "thread %lX ---------> one solution %X\n", i, ans);
          PRINT_SOL((i << K) | ans);

          res = (i << K) | ans;

          goto end;
        }
      }
    }

    if(deg1.host[K * NUM_THREADS + i] == 0)
    {
      if (check_sol(sys, (i << K) | 0, N, M) == 0)
      {
   //     LOG(INFO, "thread %lX ---------> one solution 0\n", i);
        PRINT_SOL(i << K);

        res = (i << K);

        goto end;
      }
    }
  }

end:

 // checkTime += get_ms_time();
#if 0
  // print the time for each step
  LOG(INFO, "partial ");
  LOG(INFO, "mem ");
  LOG(INFO, "kernel ");
  LOG(INFO, "recv ");
  LOG(INFO, "check #sol ");
  LOG(INFO, "(mult sol: t #ct)\n");
  LOG(INFO, "%.3f ", preTime);
  LOG(INFO, "%.3f ", memTime);
  LOG(INFO, "%.3f ", kernelTime);
  LOG(INFO, "%.3f ", recvTime);
  LOG(INFO, "%.3f ", checkTime);
  LOG(INFO, "%u ", solCount);
  LOG(INFO, "(%.3f  %u)\n", ctTime, ctCount);
#endif
  // release memory spaces
  if(sys)
  free(sys);

  return res;
}
